
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
#define MAX_VAL 4294967295LL

__global__ void matrixMul(uint16_t* matA,uint16_t* matB,uint32_t* matC,bool* existA,bool* existB,bool* existC,int n,int m,int tot){
    int elem = blockIdx.x*blockDim.x + threadIdx.x;
    if(elem<n*n){
        int x = elem/n;
        int y = elem%n;
        int i = x/m;
        int j = y/m;
        int a = x%m;
        int b = y%m;
        long long temp = 0;
        for(int k=0;k<tot;k++){
            bool A = existA[i*tot+k];
            bool B = existB[k*tot+j];
            if(A && B){
                int baseA = i*m*n + k*m*m;
                int baseB = k*m*n + j*m*m;
                for(int c=0;c<m;c++){
                    long long t1 = matA[baseA+a*m+c];
                    long long t2 = matB[baseB+c*m+b];
                    temp += t1*t2;
                }
            }
        }
        int baseC = i*m*n + j*m*m;
        long long t3 = min(MAX_VAL,temp);
        uint32_t tempC = t3;
        matC[baseC+a*m+b] = tempC;
        if(tempC!=0) existC[i*tot+j] = 1;
    }
}

int main(int argc,char **argv){
    if(argc!=4){
        cout<<"Invalid format: expected exec input1 input2 output\n";
        return 1;
    }
    int n,m,kA,kB,kC;
    FILE* inpA = fopen(argv[1],"rb");
    n = getw(inpA);
    m = getw(inpA);
    int tot = n/m;
    kA = getw(inpA);
    uint16_t *host_matA = new uint16_t[n*n]();
    uint16_t *host_matB = new uint16_t[n*n]();
    uint16_t *matA,*matB;
    hipMalloc(&matA,n*n*sizeof(uint16_t));
    hipMalloc(&matB,n*n*sizeof(uint16_t));
    uint32_t *host_matC = new uint32_t[n*n]();
    uint32_t *matC;
    hipMalloc(&matC,n*n*sizeof(uint32_t));
    bool *host_existA = new bool[tot*tot]();
    bool *host_existB = new bool[tot*tot]();
    bool *host_existC = new bool[tot*tot]();
    bool *existA,*existB,*existC;
    hipMalloc(&existA,tot*tot*sizeof(bool));
    hipMalloc(&existB,tot*tot*sizeof(bool));
    hipMalloc(&existC,tot*tot*sizeof(bool));
    for(int it=0;it<kA;it++){
        int i = getw(inpA);
        int j = getw(inpA);
        host_existA[i*tot+j] = 1;
        int t = i*m*n+j*m*m;
        for(int u = 0;u<m*m;u++){
            uint8_t low = getc(inpA);
            uint8_t upp = getc(inpA);
            host_matA[t+u] = (upp << 8) + low;
        }
    }
    hipMemcpy(existA,host_existA,tot*tot*sizeof(bool),hipMemcpyHostToDevice);
    hipMemcpy(matA,host_matA,n*n*sizeof(uint16_t),hipMemcpyHostToDevice);
    fclose(inpA);
    FILE* inpB = fopen(argv[2],"rb");
    assert(n==getw(inpB));
    assert(m==getw(inpB));
    kB = getw(inpB);
    for(int it=0;it<kB;it++){
        int i = getw(inpB);
        int j = getw(inpB);
        host_existB[i*tot+j] = 1;
        int t = i*m*n+j*m*m;
        for(int u = 0;u<m*m;u++){
            uint8_t low = getc(inpB);
            uint8_t upp = getc(inpB);
            host_matB[t+u] = (upp << 8) + low;
        }
    }
    hipMemcpy(existB,host_existB,tot*tot*sizeof(bool),hipMemcpyHostToDevice);
    hipMemcpy(matB,host_matB,n*n*sizeof(uint16_t),hipMemcpyHostToDevice);
    fclose(inpB);
    matrixMul<<<(n*n+1023)/1024,1024>>>(matA,matB,matC,existA,existB,existC,n,m,tot);
    hipDeviceSynchronize();
    hipMemcpy(host_existC,existC,tot*tot*sizeof(bool),hipMemcpyDeviceToHost);
    hipMemcpy(host_matC,matC,n*n*sizeof(uint32_t),hipMemcpyDeviceToHost);
    kC = 0;
    for(int i=0;i<tot*tot;i++){
        if(host_existC[i]) kC++;
    }
    FILE* outC = fopen(argv[3],"wb");
    putw(n,outC);
    putw(m,outC);
    putw(kC,outC);
    for(int w=0;w<tot*tot;w++){
        if(host_existC[w]){
            int i = w/tot;
            int j = w%tot;
            putw(i,outC);
            putw(j,outC);
            int base = i*m*n + j*m*m;
            for(int u=0;u<m*m;u++) putw(host_matC[base+u],outC);
        }
    }
    fclose(outC);
    hipFree(matA);
    hipFree(existA);
    hipFree(matB);
    hipFree(existB);
    hipFree(matC);
    hipFree(existC);
    return 0;
}
